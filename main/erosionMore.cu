#include "hip/hip_runtime.h"


#include <cmath>

typedef unsigned char(*pointFunction_t)(unsigned char, unsigned char);

__device__ unsigned char pComputeMin(unsigned char a, unsigned char b)
{
	return (a < b) ? a : b;
}

__device__
unsigned char pComputeMax(unsigned char a, unsigned char b)
{
	return (a > b) ? a : b;
}

template<const unsigned char boundaryValue>
__device__ void FilterStep2K(unsigned char * src, unsigned char * dst, int width, int height, int tile_w, int tile_h, const int radio, const pointFunction_t pPointOperation)
{
    extern __shared__ unsigned char smem[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

	auto x = bx * tile_w + tx;
	auto y = by * tile_h + ty - radio;

    smem[ty * blockDim.x + tx] = boundaryValue;
    __syncthreads();
    if (x >= width || y < 0 || y >= height)
	{
        return;
    }
    smem[ty * blockDim.x + tx] = src[y * width + x];
    __syncthreads();
    if (y < (by * tile_h) || y >= ((by + 1) * tile_h))
	{
        return;
    }
	auto smem_thread = &smem[(ty - radio) * blockDim.x + tx];
	auto val = smem_thread[0];
#pragma unroll
    for (auto yy = 1; yy <= 2 * radio; yy++)
	{
        val = pPointOperation(val, smem_thread[yy * blockDim.x]);
    }
    dst[y * width + x] = val;
}

template<const unsigned char boundaryValue>
__device__ void FilterStep1K(unsigned char * src, unsigned char * dst, int width, int height, int tile_w, int tile_h, const int radio, const pointFunction_t pPointOperation)
{
    extern __shared__ unsigned char smem[];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
	auto x = bx * tile_w + tx - radio;
	auto y = by * tile_h + ty;
    smem[ty * blockDim.x + tx] = boundaryValue;
    __syncthreads();
    if (x < 0 || x >= width || y >= height)
	{
        return;
    }
    smem[ty * blockDim.x + tx] = src[y * width + x];
    __syncthreads();
    if (x < (bx * tile_w) || x >= ((bx + 1) * tile_w))
	{
        return;
    }
	auto smem_thread = &smem[ty * blockDim.x + tx - radio];
	auto val = smem_thread[0];
#pragma unroll
    for (auto xx = 1; xx <= 2 * radio; xx++)
	{
        val = pPointOperation(val, smem_thread[xx]);
    }
    dst[y * width + x] = val;
}

__global__ void ErosionFilterForEachRow(unsigned char * src, unsigned char * dst, int width, int height, int tile_w, int tile_h, const int radio)
{
    FilterStep1K<255>(src, dst, width, height, tile_w, tile_h, radio, pComputeMin);
}

__global__ void ErosionFilterForEachCol(unsigned char * src, unsigned char * dst, int width, int height, int tile_w, int tile_h, const int radio)
{
    FilterStep2K<255>(src, dst, width, height, tile_w, tile_h, radio, pComputeMin);
}

void ErosionFilter(unsigned char* src, unsigned char* dst, unsigned char* temp, int width, int height, int radio)
{
	auto tile_w1 = 256, tile_h1 = 1;
	dim3 block2(tile_w1 + (2 * radio), tile_h1);
	dim3 grid2(ceil(static_cast<float>(width) / tile_w1), ceil(static_cast<float>(height) / tile_h1));

	auto tile_w2 = 4, tile_h2 = 64;
	dim3 block3(tile_w2, tile_h2 + (2 * radio));
	dim3 grid3(ceil(static_cast<float>(width) / tile_w2), ceil(static_cast<float>(height) / tile_h2));

	ErosionFilterForEachRow<<<grid2,block2,block2.y * block2.x * sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1, radio);
	auto cudaerr = hipDeviceSynchronize();

	ErosionFilterForEachCol<<<grid3,block3,block3.y * block3.x * sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2, radio);
	cudaerr = hipDeviceSynchronize();
}

__global__ void DilationFilterForEachRow(unsigned char * src, unsigned char * dst, int width, int height, int tile_w, int tile_h, const int radio)
{
    FilterStep1K<0>(src, dst, width, height, tile_w, tile_h, radio, pComputeMax);
}

__global__ void DilationFilterForEachCol(unsigned char * src, unsigned char * dst, int width, int height, int tile_w, int tile_h, const int radio)
{
    FilterStep2K<0>(src, dst, width, height, tile_w, tile_h, radio, pComputeMax);
}

void DilationFilter(unsigned char* src, unsigned char* dst, unsigned char* temp, int width, int height, int radio)
{
	auto tile_w1 = 256;
	auto tile_h1 = 1;

	dim3 block2(tile_w1 + (2 * radio), tile_h1);
	dim3 grid2(ceil(static_cast<float>(width) / tile_w1), ceil(static_cast<float>(height) / tile_h1));

	auto tile_w2 = 4;
	auto tile_h2 = 64;

	dim3 block3(tile_w2, tile_h2 + (2 * radio));
	dim3 grid3(ceil(static_cast<float>(width) / tile_w2), ceil(static_cast<float>(height) / tile_h2));

	DilationFilterForEachRow<<<grid2,block2,block2.y * block2.x>>>(src, temp, width, height, tile_w1, tile_h1, radio);
	auto cudaerr = hipDeviceSynchronize();

	DilationFilterForEachCol<<<grid3,block3,block3.y * block3.x>>>(temp, dst, width, height, tile_w2, tile_h2, radio);
	cudaerr = hipDeviceSynchronize();
}
